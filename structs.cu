
#include <hip/hip_runtime.h>
//Kyrlian, 20091121 - 20091201
//////////////////////////// TYPEDEF ////////////////////////////
typedef struct mlight{
 float3 dir;
 uint3 colour;
} mlight;

typedef struct __align__(16) observ{
 float3 point;
 float3 axis;
 float3 x;
 float3 y;
 float focal;
 float width;
 float scale;
 float maxdist;
 int maxiter;
 uint3 ambient;//ambient light comes from everywhere and is reflected to all direction
 uint3 diffuse;//diffuse light comes from everywhere and is reflected according to normal of surface
 mlight light;
 int power;
 float epsilonfactor;
} observ;
/// EOF ///
