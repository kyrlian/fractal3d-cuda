#include "hip/hip_runtime.h"
//Kyrlian, 20091121 - 20091201
// adjust with higher blocksize - or with smaller as threads might wait a lot for each other (lockstep)
#define BLOCKSIZE 8//depends on memory needs: 4-8 is ok, 16 is not - use profiler to estimate
//Tested :
//4*4   threads per block (16) : 618ms, 
//6*6   threads per block (36) : 510ms
//8*8   threads per block (64) : 334ms, 
//10*10 threads per block (100): 376ms, - I guess pow(2) are better for memory management
//12*12 threads per block (144): 446ms,
//16*16 threads per block (256): 338ms. - not as good as 8*8, maybe because of lockstep
//17*17 doesnt run : too much ressources
//////////////////////////// KERNEL ////////////////////////////
__global__ void cudakernel(const int screenw, const int screenh, const observ* view, uchar4* pixeltable){
//use float4 for device for faster access, use float3 for local or shared
 int scx = blockIdx.x * blockDim.x + threadIdx.x;
 int scy = blockIdx.y * blockDim.y + threadIdx.y;
 // register __shared__ variables, should be quicker than going to view, which is in device global memory
 __shared__ float3 point, vx, vy, vfocal, lightdir;
 __shared__ float halfw, halfh, scale, maxdist, epsilonfactor;
 __shared__ int maxiter, power;
 __shared__ uint3 ambient, diffuse, lightcolour;
// if(scx < screenw && scy < screenh){//this is just to check and avoid outbound access
  if ((threadIdx.x==0) && (threadIdx.y==0)){//shared variables - shared among threads of the block
   point    = view->point;
   vx       = view->x;
   vy       = view->y;
   scale    = view->scale;
   maxdist  = view->maxdist;
   maxiter  = view->maxiter;
   power    = view->power;
   epsilonfactor = view->epsilonfactor;
   ambient  = view->ambient;
   diffuse  = view->diffuse;
   lightdir = view->light.dir;
   lightcolour = view->light.colour;
   vfocal = view->axis * view->focal;
   halfh = screenh/2;
   halfw = screenw/2;
  }
  __syncthreads();
  float3 raydir = normalize(vfocal + vx * (scx-halfw)*scale + vy * (scy-halfh)*scale);//is raydir device or local ? should be local I guess...
//  pixeltable[scy * screenw + scx] = RayTrace(view, raydir);//view is in device global memory, should use shared memory more (faster)
  pixeltable[scy * screenw + scx] = RayTrace(point, scale, maxdist, maxiter, power, epsilonfactor, ambient, diffuse, lightdir, lightcolour, raydir);
//  }
}
//////////////////////////// RUNNER (main) ////////////////////////////
__host__ void cuda_draw(){
 cutResetTimer(hTimer);
 //update scale (in case width was changed or image size was changed)
  view.scale = view.width/imageW;
 // map dst to opengl buffer 
  cutilSafeCall(cudaGLMapBufferObject((void**)&d_dst, gl_PBO));//map the PBO - locks it from opengl
 // copy to device memory
  cutilSafeCall(hipMemcpy(d_view, &view, sizeof(observ), hipMemcpyHostToDevice) );        
 // setup execution parameters
  dim3 block(BLOCKSIZE, BLOCKSIZE);
  dim3 grid(imageW/BLOCKSIZE, imageH/BLOCKSIZE); //thus w and h must be BLOCKSIZE multiples.
 // execute the kernel
  cudakernel<<< grid, block >>>(imageW, imageH, d_view, d_dst);
 // check if kernel execution generated and error
  cutilCheckMsg("Kernel execution failed");
 //make sure all kernels are done
  hipDeviceSynchronize();
 // unmap dst from opengl buffer  
  cutilSafeCall(cudaGLUnmapBufferObject(gl_PBO));//free the PBO so we can draw it
 //set up colors TODO see what this does exactly...
    glBindTexture(GL_TEXTURE_2D, gl_Tex);
   	glTexSubImage2D(GL_TEXTURE_2D, 0, 0, 0, imageW, imageH, GL_RGBA, GL_UNSIGNED_BYTE, BUFFER_DATA(0));
    // fragment program is required to display floating point texture
 //  glBindProgramARB(GL_FRAGMENT_PROGRAM_ARB, gl_Shader);
 //  glEnable(GL_FRAGMENT_PROGRAM_ARB);
 //  glDisable(GL_DEPTH_TEST);
    glBegin(GL_QUADS);
      glTexCoord2f(0.0f, 0.0f); glVertex2f(0.0f, 0.0f);
      glTexCoord2f(1.0f, 0.0f); glVertex2f(1.0f, 0.0f);
      glTexCoord2f(1.0f, 1.0f); glVertex2f(1.0f, 1.0f);
      glTexCoord2f(0.0f, 1.0f); glVertex2f(0.0f, 1.0f);
    glEnd();
    glBindTexture(GL_TEXTURE_2D, 0);
  //  glDisable(GL_FRAGMENT_PROGRAM_ARB);
 // really displays the image
  glutSwapBuffers();
 //print timer
  printf("GPU time : %.0f ms\n", cutGetTimerValue(hTimer));
}
