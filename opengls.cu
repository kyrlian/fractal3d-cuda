#include "hip/hip_runtime.h"
//Kyrlian, 20091121 - 20091201
//////////////////// OpenGL init //////////////////////
void initGL(int argc, char **argv){
 printf("initGL:\n");
 printf("Initializing GLUT...\n");
  glutInit(&argc, argv);
  glutInitDisplayMode(GLUT_RGBA | GLUT_DOUBLE);
  glutInitWindowSize(imageW, imageH);
  glutInitWindowPosition(0, 0);
  glutCreateWindow(argv[0]);
  glutSetWindowTitle(windowsTitle);
 printf("Loading extensions: %s\n", glewGetErrorString(glewInit()));
 if (!glewIsSupported( "GL_VERSION_1_5 GL_ARB_vertex_buffer_object GL_ARB_pixel_buffer_object" )) {
  fprintf(stderr, "Error: failed to get minimal extensions for demo\n");
  fprintf(stderr, "This sample requires:\n");
  fprintf(stderr, "  OpenGL version 1.5\n");
  fprintf(stderr, "  GL_ARB_vertex_buffer_object\n");
  fprintf(stderr, "  GL_ARB_pixel_buffer_object\n");
  exit(-1);
 }
 printf("OpenGL window created.\n");
}
//////////////////// OpenGL shaders //////////////////////
// gl_Shader for displaying floating-point texture
/*
static const char *shader_code = 
"!!ARBfp1.0\n"
"TEX result.color, fragment.texcoord, texture[0], 2D; \n"
"END";

GLuint compileASMShader(GLenum program_type, const char *code)
{
    GLuint program_id;
    glGenProgramsARB(1, &program_id);
    glBindProgramARB(program_type, program_id);
    glProgramStringARB(program_type, GL_PROGRAM_FORMAT_ASCII_ARB, (GLsizei) strlen(code), (GLubyte *) code);
    GLint error_pos;
    glGetIntegerv(GL_PROGRAM_ERROR_POSITION_ARB, &error_pos);
    if (error_pos != -1) {
        const GLubyte *error_string;
        error_string = glGetString(GL_PROGRAM_ERROR_STRING_ARB);
        fprintf(stderr, "Program error at position: %d\n%s\n", (int)error_pos, error_string);
        return 0;
    }
    return program_id;
}*/
//////////////////// OpenGL init buffers //////////////////////
void initOpenGLBuffers(int w, int h){
    printf("initOpenGLBuffers:\n");
    // delete old buffers
    if (h_Src) {
        free(h_Src);
        h_Src = 0;
    }
    if (gl_Tex) {
        glDeleteTextures(1, &gl_Tex);
        gl_Tex = 0;
    }
    if (gl_PBO) {
        cudaGLUnregisterBufferObject(gl_PBO);
        glDeleteBuffers(1, &gl_PBO);
        gl_PBO = 0;
    }
    // check for minimized window
    if ((w==0) && (h==0)) {
        return;
    }
    // allocate new buffers
	h_Src = (uchar4*)malloc(w * h * 4);
    printf("Creating GL texture...\n");
        glEnable(GL_TEXTURE_2D);
        glGenTextures(1, &gl_Tex);
        glBindTexture(GL_TEXTURE_2D, gl_Tex);
        glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_CLAMP);
        glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_CLAMP);
        glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);
        glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
        glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA8, w, h, 0, GL_RGBA, GL_UNSIGNED_BYTE, h_Src);
    printf("Texture created.\n");
    printf("Creating PBO...\n");
        glGenBuffers(1, &gl_PBO);
        glBindBuffer(GL_PIXEL_UNPACK_BUFFER_ARB, gl_PBO);
        glBufferData(GL_PIXEL_UNPACK_BUFFER_ARB, w * h * 4, h_Src, GL_STREAM_COPY);
        //While a PBO is registered to CUDA, it can't be used 
        //as the destination for OpenGL drawing calls.
        //But in our particular case OpenGL is only used 
        //to display the content of the PBO, specified by CUDA kernels,
        //so we need to register/unregister it only once.
        cutilSafeCall( cudaGLRegisterBufferObject(gl_PBO) );
    printf("PBO created.\n");
    // load shader program
 //   gl_Shader = compileASMShader(GL_FRAGMENT_PROGRAM_ARB, shader_code);
}
//////////////////// OpenGL reshape //////////////////////
void reshapeFunc(int w, int h){
    printf("reshapeFunc\n");
    w=BLOCKSIZE*(int)(w/BLOCKSIZE);// all sizes must be multiples of BLOCKSIZE
    h=BLOCKSIZE*(int)(h/BLOCKSIZE);
    glViewport(0, 0, w, h);
    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();
    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();
    glOrtho(0.0, 1.0, 0.0, 1.0, 0.0, 1.0);
    initOpenGLBuffers(w, h);
    imageW = w;
    imageH = h;
}
//////////////////// OpenGL cleaner //////////////////////
void cleanup(){
 printf("OpenGL Cleanup\n");
 if (h_Src) {
  free(h_Src);
  h_Src = 0;
 }
 cutilSafeCall(hipFree(d_view));//free d_view
 cutilCheckError(cutStopTimer(hTimer) );
 cutilCheckError(cutDeleteTimer(hTimer));
 cutilSafeCall(cudaGLUnregisterBufferObject(gl_PBO));
 glBindBuffer(GL_PIXEL_UNPACK_BUFFER_ARB, 0);
 glDeleteBuffers(1, &gl_PBO);
 glDeleteTextures(1, &gl_Tex);
 glDeleteProgramsARB(1, &gl_Shader);
 if (g_FrameBufferObject) {
  delete g_FrameBufferObject; g_FrameBufferObject = NULL;
 }
 if (g_CheckRender) {
  delete g_CheckRender; g_CheckRender = NULL;
 }
}
//////////////////  OpenGL keyboard function ////////////////// 
void keyboardFunc(unsigned char k, int, int){
 switch (k){
  case '\033':
  case 'a':
  case 'A':
   printf("bye!\n");
   exit(0);
   break;
//DRAW
  case ' '://just redraw
   break;
/*   
  case 'p'://save bmp
  case 'b'://save bmp
   savebmp();
  //break;//dont break, wait for the printinfo part
*/
  case 'o'://output view info to stdout
   printinfo();
  break;
//ITERATIONS
  case 'w':
   view.maxiter--;
   printf("maxiter: %i\n",view.maxiter);
  break;
  case 'W':
   view.maxiter++;
   printf("maxiter: %i\n",view.maxiter);   
  break;
//EPSILON FACTOR
  case 'x':
   view.epsilonfactor *= 0.1f;
   printf("epsilonfactor: %f\n",view.epsilonfactor);
  break;
  case 'X':
   view.epsilonfactor *= 10.0f;
   printf("epsilonfactor: %f\n",view.epsilonfactor);   
  break;
//POWER p/P  
  case 'p':
   view.power--;
   printf("power: %i\n",view.power);
  break;
  case 'P':
   view.power++;
   printf("power: %i\n",view.power);   
  break;
//MOVEMENT (zqsd, ec) azerty
  case 'z'://forward
   view.point+=view.axis * movespeed;
   float3_print((char*)"view point",view.point);
  break;
  case 's'://backward
   view.point-=view.axis * movespeed;
   float3_print((char*)"view point",view.point);
  break;
  case 'q'://left
   view.point-=view.x * movespeed;
   float3_print((char*)"view point",view.point); 
  break;
  case 'd'://right
     view.point+=view.x * movespeed;
     float3_print((char*)"view point",view.point); 
  break;
  case 'e'://up
     view.point+=view.y * movespeed;
     float3_print((char*)"view point",view.point); 
  break;
  case 'c'://down
     view.point-=view.y * movespeed;       
     float3_print((char*)"view point",view.point);
  break;
//WIDTH (r/v)
  case 'r'://zoom in
   view.width-= movespeed;
   printf("width: %f\n",view.width);
  break;
  case 'v'://zoom out
   view.width+= movespeed;
   printf("width: %f\n",view.width);   
  break;
//LOOK (4568) and circle around (ijkl)
  case 'k'://change axis and y and point
   rotate(&(view.point), view.x, moveangle);//dont break, we need to look down
  case '8'://change axis and y
   rotate(&(view.axis), view.x, moveangle);
   rotate(&(view.y), view.x, moveangle);
  break;
  case 'i'://change axis and y and point
   rotate(&(view.point), view.x, -1*moveangle);//dont break, we need to look down
  case '5'://change axis and y
   rotate(&(view.axis), view.x, -1*moveangle);
   rotate(&(view.y), view.x, -1*moveangle);
  break;
  case 'l'://change axis and x and point
   rotate(&(view.point), view.y, -1*moveangle);//dont break, we need to look down
  case '4'://change axis and x
   rotate(&(view.axis), view.y, -1*moveangle);
   rotate(&(view.x), view.y, -1*moveangle);
  break;
  case 'j'://change axis and x and point
   rotate(&(view.point), view.y, moveangle);//dont break, we need to look down
  case '6'://change axis and x
   rotate(&(view.axis), view.y, moveangle);
   rotate(&(view.x), view.y, moveangle);
  break;
  case '7'://change y and x
   rotate(&(view.x), view.axis, -1*moveangle);
   rotate(&(view.y), view.axis, -1*moveangle);
  break;
  case '9'://change y and x
   rotate(&(view.x), view.axis, moveangle);
   rotate(&(view.y), view.axis, moveangle);   
  break;
//move light (g/h)
  case 'g'://rotate light around view vertical axis
   rotate(&(view.light.dir), view.y, moveangle);
  break;
  case 'h':
   rotate(&(view.light.dir), view.y, -1*moveangle);
  break;
//OTHER
/*  case 'f'://toggle full screen - no need, use window manager to do this
   static int fullscreen = 0;
   fullscreen=1-fullscreen;
   if (fullscreen==1)glutFullScreen();
   else glutReshapeWindow(INIW,INIH);
  break;*/
 default://display usage
    usage();
   break;
 }
 //whatever key was pressed :
  // normalize all axes, to avoid rotation crunch
   view.axis = normalize(view.axis);
   view.x= normalize(view.x);
   view.light.dir = normalize(view.light.dir);
   view.y = normalize(cross(view.axis,view.x));
  // autofocus to compute maxdist
   float a=autofocus();
   if(a>0){
    view.maxdist = 10 * a;
    movespeed = a * view.width * 0.1f;
    moveangle = fminf(atan2(a, view.width),0.1f);
   }
  // call for redraw
  glutPostRedisplay(); //glut will call the redisplay, wich will trigger a redraw
}
////////////////// OpenGL mouse click function ////////////////// 
void clickFunc(int button, int state, int x, int y){
 printf("click\n");
 if (button == 0) leftClicked = !leftClicked;
 if (button == 1) middleClicked = !middleClicked;
 if (button == 2) rightClicked = !rightClicked;
 int modifiers = glutGetModifiers(); 
 if (leftClicked && (modifiers & GLUT_ACTIVE_SHIFT)) {
  leftClicked = 0;
  middleClicked = 1;
 }
 if (state == GLUT_UP) {
  leftClicked = 0;
  middleClicked = 0;
 }
 lastx = x;
 lasty = y;
}
////////////////// OpenGL mouse motion function /////////////////////
void motionFunc(int x, int y){
// double fx = (double)(x - lastx) / 50.0 / (double)(imageW);        
// double fy = (double)(lasty - y) / 50.0 / (double)(imageH);
} // motionFunc
////////////////// OpenGL idle function /////////////////////
void idleFunc(){
//glutPostRedisplay();//don't - this would mean redraw all the time
}
