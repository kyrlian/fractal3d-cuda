#include "hip/hip_runtime.h"
//Kyrlian, 20091121 - 20091201
 #include <stdio.h>
 #include <stdlib.h>
 #include <string.h>
 #include <math.h>
 #include <cstdio>
// opengl
 #include <GL/glew.h>
 #include <GL/glut.h>
#ifdef _WIN32
#include <GL/wglew.h>
#endif 
// cuda
#include <hip/hip_runtime.h>//_api.h>
 #include <cutil_inline.h>
 #include <cutil_gl_inline.h>
 #include <cutil_math.h>
 #include <cuda_gl_interop.h>
 #include <rendercheck_gl.h>
// struct include (observ and light)
 #include "structs.cu"
// some constants - other constants are in each code file
// #define BITSPERPIXEL 32
 #define BUFFER_DATA(i) ((char *)0 + i) // used in cuda_draw for textures
//////////////////////////// GLOBAL ////////////////////////////
//OpenGL PBO and texture "names"
 GLuint gl_PBO, gl_Tex, gl_Shader;
 //Source image on the host side
 uchar4 *h_Src = 0;
 // Destination image on the GPU side
 uchar4 *d_dst = NULL;
 //Original image width and height
 #define INIW 800
 #define INIH 600
 int imageW = INIW, imageH = INIH;
 // User interface variables
 int lastx = 0;
 int lasty = 0;
 bool leftClicked = false;
 bool middleClicked = false;
 bool rightClicked = false;
 // CheckFBO/BackBuffer class objects
 CFrameBufferObject *g_FrameBufferObject = NULL;
 CheckRender        *g_CheckRender = NULL;
 // windows title
 static char *windowsTitle = "CUDA fractal3d";
 // Timer ID
 unsigned int hTimer;
 // view is global (easier)
 observ view;
 observ* d_view; //pointer for device access
 //movement
 float movespeed = 0.1f;
 float moveangle = 0.1f;
 //misc
 //int finished = 0; 
//////////////////////////// Project includes //////////////////////////// 
// some globals are used in there, that's why I waited till here. Did I mention this is not good practice and I should use headers ?
 #include "misc.cu"
 #include "raytrace.cu"
 #include "cuda_draw.cu"
 #include "opengls.cu"
//////////////////////////// MAIN ////////////////////////////
int main(int argc, char* argv[]){
 // init light
  mlight viewlight;
  viewlight.dir=normalize(make_float3(-1,1,1));
  viewlight.colour=make_uint3(255, 215, 0);
 //init view
  view.point   = make_float3(0,-3,0);
  view.axis    = normalize(make_float3(0,1,0));
  view.x       = normalize(make_float3(1,0,0));
  view.y       = normalize(cross(view.axis,view.x));  
  view.focal   = 3;//focal(fixed)
  view.width   = 4;//width(user mod)
  view.scale   = view.width/imageW;//scale(computed in Draw)
  view.maxdist = 2*length(view.point);//maxdist(computed from autofocus)
  view.maxiter = 10;
  view.ambient = make_uint3(5);
  view.diffuse = make_uint3(40);
  view.light   = viewlight; 
  view.power   = 8;
  view.epsilonfactor = 0.1f;
 // Initialize OpenGL context first before the CUDA context is created.  This is needed
 // to achieve optimal performance with OpenGL/CUDA interop.
  initGL( argc, argv );
 // choose and init opengl cuda device
  cutilChooseCudaGLDevice(argc, argv);
  //allocate memory on the device for dview
  cutilSafeCall(hipMalloc((void**) &d_view, sizeof(observ)));  
  //display usage
  usage();
 //opengl mapping to my functions
  glutDisplayFunc(cuda_draw);//opengl will call this when needed
  //glutIdleFunc(idleFunc);//this seems to take all cpu - we do nothing when idle, hus uneeded
  glutKeyboardFunc(keyboardFunc);
  glutMouseFunc(clickFunc);
  glutMotionFunc(motionFunc);
  glutReshapeFunc(reshapeFunc);//this is called when the windows is created, the opengl/cuda buffers are created then.
 //timer
  cutilCheckError(cutCreateTimer(&hTimer));
  cutilCheckError(cutStartTimer(hTimer));
 //loop
  atexit(cleanup);//cleanup function
  glutMainLoop();//main loop
 //end
 hipDeviceReset();//cuda exit
 cutilExit(argc, argv);
 return 0;
}
