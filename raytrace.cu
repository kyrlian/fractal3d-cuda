#include "hip/hip_runtime.h"
//Kyrlian, 20091121 - 20091201
//fractal formulas from http://www.fractalforums.com/3d-fractal-generation/true-3d-mandlebrot-type-fractal/?action=printpage
//#define EPSILON 0.000001f // is now adaptative (=scale)
#define DIVERGENCE 2.0f//float
//#define POWER 8.0f //float
#define PHONG_EXP 8//
//#define EPSILONFACT 0.1f//epsilon=scale*EPSILONFACT
//////////////////////////// RAY STEP / TEST ////////////////////////////
// compute raytrace step at given point - scalar derivative
__host__ float HRayStep(const float3 &point, int maxiter, int power){
 float3 z = point;
 float  r = length(z);
 float dr = 1.0f;
 int    i = maxiter;                   //max iteration count
 while(r<DIVERGENCE && i--) {
  float ph = asinf( z.z/r );
  float th = atan2f( z.y,z.x );
  float zr = powf( r, power - 1.0f );
  dr = zr * dr * power + 1.0f;
  zr *= r;
  float sph,cph; sincosf(power*ph, &sph, &cph);
  float sth,cth; sincosf(power*th, &sth, &cth);
  z.x = zr * cph*cth + point.x;
  z.y = zr * cph*sth + point.y;
  z.z = zr * sph     + point.z;
  r=length(z);
 }
  return 0.5f*logf(r)*r/dr;
}
//device version uses some specific device functions, runtime si divided by 2 using cuda __ functions !
inline __device__ float RayStep(const float3 &point, int maxiter, int power){
 float3 z = point;
 float  r = length(z);
 float dr = 1.0f;
 int    i = maxiter;                   //max iteration count
 while(r<DIVERGENCE && i--) {
  float ph = asinf( __fdividef( z.z, r ) );
  float th = atan2f( z.y, z.x );
  float zr = __powf( r, power - 1.0f );
  dr = zr * dr * power + 1.0f;
  zr *= r;
  float sph,cph; __sincosf(power*ph, &sph, &cph);
  float sth,cth; __sincosf(power*th, &sth, &cth);
  z.x = zr * cph*cth + point.x;
  z.y = zr * cph*sth + point.y;
  z.z = zr * sph     + point.z;
  r=length(z);
 }
  return 0.5f * __logf(r) * __fdividef (r , dr);
}
// Test a point against fractal iteration
/*
inline __device__ int RayTest(const float3 &point, int maxiter, int power){ //return 0 if not found or 1 if found
 //test:draw square (normal is not computed from this, so shadows will be weird) : 
 //float c=0.5f; if (fabs(point->x)<c && fabs(point->y)<c && fabs(point->z)<c)return 1;else return 0;
 float3 z = make_float3(0.0f);
 float  r = 1e-10;
 int    i = maxiter;
 while( i-- ) {
  float ph = asinf( __fdividef( z.z, r ) );
  float th = atan2f( z.y, z.x );
  float zr = __powf(r, power);
  float sph,cph; __sincosf(power*ph, &sph, &cph);
  float sth,cth; __sincosf(power*th, &sth, &cth);
  z.x = zr * cph*cth + point.x;
  z.y = zr * cph*sth + point.y;
  z.z = zr * sph     + point.z;
  r=length(z);
  if( r > DIVERGENCE )return 0;
 }
 return 1;
}
*/
//////////////////////////// NORMAL ////////////////////////////
//returns a vector normal to the surface, inbound
inline __device__ float3 RayNormal(float3 &point, int maxiter, int power, const float eps=1e-3f){//quicker: 30ms, less right
   float3 t = point;
   float c=RayStep(t, maxiter, power);
                       t.x+=eps;
   float cx = RayStep( t, maxiter, power );
   t = point;          t.y+=eps;
   float cy = RayStep( t, maxiter, power );
   t = point;          t.z+=eps;
   float cz = RayStep( t, maxiter, power );
   return normalize(make_float3( c-cx, c-cy, c-cz ));
}
/*
inline __device__ float3 RayNormal(const float3 &point, int maxiter, int power, const float eps=1e-3f){
   float cx = RayStep(point - make_float3(eps,0,0), maxiter, power) - RayStep(point + make_float3(eps,0,0), maxiter, power);
   float cy = RayStep(point - make_float3(0,eps,0), maxiter, power) - RayStep(point + make_float3(0,eps,0), maxiter, power);
   float cz = RayStep(point - make_float3(0,0,eps), maxiter, power) - RayStep(point + make_float3(0,0,eps), maxiter, power);
   return normalize(make_float3( cx, cy, cz ));
}*/
//////////////////////////// RAY MARCHER (A/NA) ////////////////////////////
// adaptative raymarch - uses raystep
// return 1 if found 0 if not, set point coords in last param
__host__ int HRayMarchA(const float3 &from, const float3 &direction, int power, float epsilon, float viewlimit, int maxiter, float3 *point, float *dist){
 *point = from;
 *dist  = 0.0f;
 while(*dist<viewlimit){
  float step = HRayStep(*point, maxiter, power);
  if(step < epsilon)return 1;// exit is close enough to point - ie found
  *point += step * direction;
  *dist  += step;
 }
 return 0;
}
inline __device__ int RayMarchA(const float3 &from, const float3 &direction, int power, float epsilon, float viewlimit, int maxiter, float3 *point, float *dist){
 *point = from;
 *dist  = 0.0f; 
 while(*dist<viewlimit){
  float step = RayStep(*point, maxiter, power);
  if(step < epsilon)return 1;// exit is close enough to point - ie found
  *point += step * direction;
  *dist  += step;  
 }
 return 0;
}

// non adaptative raymarch - uses (raytest) RayStep to have the same result as RayMarchA for in/out definition, except we walk at a non adaptative step
// return 1 if found 0 if not, set point coords in last param
inline __device__ int RayMarchNA(const float3 &from, const float3 &direction, int power, float step, float epsilonfactor, float viewlimit, int maxiter, float3 *point, float *dist){
 *point = from;
 *dist  = 0.0f;  
 float epsilon = step * epsilonfactor;
 while(*dist<viewlimit){
  *point += step * direction;
  *dist  += step;
  step *= 1.1f;//increase step regularly (ie resolution decrease with distance)
//  if (RayTest(*point, maxiter)==1) return 1; //exit if point found
  if (RayStep(*point, maxiter, power)< epsilon) return 1;//use same test as RayMarchA, except our step is fixed
 }
 return 0;
}
//////////////////////////// AUTOFOCUS ////////////////////////////
// returns distance to point in center of the screen, used to compute view max dist (10*autofocus)
__host__ float autofocus(){
 float3 found;
 float founddist;
 if(HRayMarchA(view.point, view.axis, view.power, view.scale * view.epsilonfactor , view.maxdist, view.maxiter, &found, &founddist)==1)
  return founddist;
 else return 0.0f;
}
//////////////////////////// CUSTOM float*uint3 OPERATOR ////////////////////////////
inline __device__ uint3 operator*(float s, uint3 a)//this operator doenst exist in cuda_math and I need it below
{    return make_uint3(a.x * s, a.y * s, a.z * s);   }
inline __device__ uint3 operator*(uint3 a, float s)//define both orders
{    return make_uint3(a.x * s, a.y * s, a.z * s);   }
//////////////////////////// RAY TRACER (main) ////////////////////////////
// returns pixel color
//all args are shared in VRAM, thus const
//__device__ uchar4 RayTrace(const observ* view, const float3 &raydir){
__device__ uchar4 RayTrace(const float3 &point, const float &scale, const float &maxdist, const int &maxiter, const int &power, const float &epsilonfactor, const uint3 &gambient, const uint3 &gdiffuse, const float3 &lightdir, const uint3 &lightcolour, const float3 &raydir){
 uint3 c = make_uint3(0);
 float3 found, found2;
 float founddist, founddist2;
 //we set epsilon as scale/12, which seems to be fine
 if(RayMarchA(point, raydir, power, scale * epsilonfactor, maxdist, maxiter, &found, &founddist)==1){
  float3 fnormal = RayNormal(found, maxiter, power, scale * epsilonfactor);
  float  dp = __saturatef(dot(fnormal, raydir));
// float  fact = (1.0f- __fdividef( founddist, viewlimit ) );//so attenuation is proportional to distance and max distance - bad if maxdistance is adaptative
  c = gambient + dp * gdiffuse;//this is called the 'ambient' part, ie touched by no light. I added a diffuse part for shadowing.
// this would be done for each light in the scene
  if(RayMarchNA(found, -1.0f*lightdir, power, scale, epsilonfactor, maxdist, maxiter, &found2, &founddist2)==0){//we can deactivate this test to save some time, meaning no real light casting and just simulate shadow with normals.
   // in light, compute diffuse and specular light amounts
   float diffuse = __saturatef(dot(fnormal, lightdir));// staturate clamps in 0-1
//   float specular = __powf (__saturatef(dot(reflect(raydir, fnormal), lightdir)), PHONG_EXP);
   float specular = __powf (__saturatef(dot((lightdir+raydir)/2, fnormal)), PHONG_EXP);//quicker by 3ms
   c += diffuse * lightcolour + specular * make_uint3(255);//diffuse reflection is of light colour, and specular reflexion is white
  }
  c = min( c, make_uint3(255) );//phong = ambient + diffuse + specular
  // we could add Ambient occlusion (trace in every direction, add light for each not-blocked ray
 }
 return make_uchar4(c.x,c.y,c.z,1);
}
