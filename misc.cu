#include "hip/hip_runtime.h"
//Kyrlian, 20091121 - 20091201
//////////////////////////// USAGE ////////////////////////////
void usage(){
 printf("Usage :\n");
// printf("  RETURN     : re-draw and save\n");
 printf("  z, q, s, d : move\n");
 printf("  e, c       : strafe up/down\n");
 printf("  r, v       : zoom in/out\n");
 printf("  i, j, k, l : rotate view around origin\n");
 printf("  4, 5, 6, 8: look around\n");
 printf("  g, h       : rotate light around view vertical axis\n");
 printf("  w, W       : decrease, increase maxiter\n");
// printf("  f          : toggle full screen\n");
// printf("  p, b       : save bmp\n");
 printf("  o          : print position\n");
 printf("  u          : print (this) usage\n");
 printf("  a          : quit\n");
 printf("Display is refreshed after any keystroke.\n");
 fflush(stdout);     
}
//////////////////////////// INFO ////////////////////////////
void float3_print(char *name, const float3 a){
 printf("%s: %f,%f,%f\n", name, a.x, a.y, a.z);
}
void printinfo(){
 printf("-------------------------------------\n");
 float3_print((char*)"view point",view.point);
 float3_print((char*)"view axis ",view.axis);
 float3_print((char*)"view x    ",view.x);
 float3_print((char*)"view y    ",view.y);
 float3_print((char*)"light dir ",view.light.dir);
              printf("focal     : %f\n",view.focal);
              printf("width     : %f\n",view.width);
              printf("scale     : %f\n",view.scale);
              printf("maxdist   : %f\n",view.maxdist);
              printf("maxiter   : %i\n",view.maxiter);
 printf("%s: %i,%i,%i\n", (char*)"ambient     ", view.ambient.x, view.ambient.y, view.ambient.z);
 printf("%s: %i,%i,%i\n", (char*)"diffuse     ", view.diffuse.x, view.diffuse.y, view.diffuse.z);
 printf("%s: %i,%i,%i\n", (char*)"light colour", view.light.colour.x, view.light.colour.y, view.light.colour.z);
 printf("-------------------------------------\n");
 fflush(stdout);
}
//////////////////////////// BMP ////////////////////////////
void savebmp(){
 static int i=0;
 char filename [20];
 sprintf(filename, "out_%d.bmp", i++);
 //SDL_SaveBMP(screen, filename);//TODO : well, opengl doesnt have a simple function to save an image :(
 printf("-------------------------------------\n");
 printf("Saving view to %s, parameters follow:\n",filename);
}
//////////////////////////// ROTATE ////////////////////////////
void rotate(float3* vect, const float3 axis, const float a){
 float x=vect->x; float y=vect->y; float z=vect->z;
 float u=axis.x;  float v=axis.y;  float w=axis.z;
 float ux=u*x; float uy=u*y; float uz=u*z;
 float vx=v*x; float vy=v*y; float vz=v*z;
 float wx=w*x; float wy=w*y; float wz=w*z;
 float sa, ca; sincosf(a, &sa, &ca);
 vect->x=u*(ux+vy+wz)+(x*(v*v+w*w)-u*(vy+wz))*ca+(-wy+vz)*sa;
 vect->y=v*(ux+vy+wz)+(y*(u*u+w*w)-v*(ux+wz))*ca+(wx-uz)*sa;
 vect->z=w*(ux+vy+wz)+(z*(u*u+v*v)-w*(ux+vy))*ca+(-vx+uy)*sa;
}
/// EOF ///
